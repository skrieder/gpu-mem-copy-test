
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int host_a = 2;

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  hipStream_t stream;
  hipStreamCreate(&stream);

  
  // loop over the loop count and copy to device
  for(int i = 0; i < loopCount; i++){
    int *dev_a;
    
    // malloc on the device
    hipMalloc((void **)&dev_a, size);
  
    //    cudaMemcpyAsync(dev_a, &host_a, size, cudaMemcpyHostToDevice, stream);
    
    // cudaError_t e = cudaStreamSynchronize(stream);
    //if( e!=cudaSuccess)printf("%s\n", cudaGetErrorString(e));
  }


  // free device memory
  //cudaFree(dev_a);

  // return with no errors
  return 0;
}
