
#include <hip/hip_runtime.h>
#include <stdio.h>

//__global__ void kernel( void ) {
  // does nothing
//}

int main(int argc, char** argv) {
  
  // default the loop count to equal 1
  int loopCount = 1;

  // take in a command line arg to set the loop count
  if(argc > 1){
    loopCount = atoi(argv[1]);
  }

  // delcare two variables
  int host_a;
  int *dev_a;

  hipStream_t stream;
  hipStreamCreate(&stream);

  // get the size of an int for the cuda malloc
  int size = sizeof(int);

  // malloc on the device
  hipMalloc((void **)&dev_a, size);

  // copy over to device
  hipMemcpy(dev_a, &host_a, size, hipMemcpyHostToDevice);

  // call an empty kernel
  //kernel<<<1,1>>>();

  // loop over the loop count and copy to host
  for(int i = 0; i < loopCount; i++){
    hipMemcpyAsync(&host_a, dev_a, size, hipMemcpyDeviceToHost, stream);
  }

  hipError_t e = hipStreamSynchronize(stream);
  if( e!=hipSuccess)printf("%s\n", hipGetErrorString(e));

  // free device memory
  hipFree(dev_a);

  // return with no errors
  return 0;
}
